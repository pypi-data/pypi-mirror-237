#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include "diamond.h"
#include "field.h"
#include "testutils.h"

namespace field {
namespace {

using defs::IsAux;
using diamond::X;
using diamond::Y;
using diamond::Z;

// Just a simple point test.
TEST(Field, Field) {
  XY domain(10, 10);
  int npml = 5;
  int zshift = 10;
  int nout = 4;
  defs::RunShape::Vol sub(N, domain.x - N, N, domain.y - N, 0,
                          diamond::ExtZz<float>(npml));

  testutils::Array<int> arr(ExternalElems<int>(sub, nout, npml));
  Cell<int> cell;
  InitCell(cell, 0);
  cell.Set(42, Node(0, 0, 1, diamond::E, diamond::X));
  XY pos(5, 5);
  int threadpos(3);
  int outindex = 2;
  WriteCell(cell, arr.Ptr(), pos, outindex, threadpos, domain, npml, zshift,
            /*isaux=*/false, sub);
  EXPECT_EQ(
      arr[ExtNodeIndex(Node(5, 5, 51, diamond::E, diamond::X), outindex, sub)],
      42);
}

TEST(Field, FieldHalf2) {
  XY domain(10, 10);
  int nout = 4;
  int npml = 5;
  int zshift = 7;
  defs::RunShape::Vol sub(N, domain.x - N, N, domain.y - N, 0,
                          diamond::ExtZz<half2>(npml));

  testutils::Array<float> arr(ExternalElems<half2>(sub, nout, npml));
  Cell<half2> cell;
  InitCell(cell, defs::Zero<half2>());
  cell.Set(__floats2half2_rn(42.0f, 43.0f),
           Node(0, 0, 1, diamond::E, diamond::X));
  XY pos(5, 5);
  int threadpos(1);
  int outindex = 2;
  WriteCell(cell, arr.Ptr(), pos, outindex, threadpos, domain, npml, zshift,
            /*isaux=*/false, sub);
  EXPECT_EQ(
      arr[ExtNodeIndex(Node(5, 5, 106, diamond::E, diamond::X), outindex, sub)],
      42.0f);
  // Note that the lo-hi values of the half2 are on "opposite" sides of the
  // wrap-around.
  EXPECT_EQ(
      arr[ExtNodeIndex(Node(5, 5, 0, diamond::E, diamond::X), outindex, sub)],
      43.0f);
}

} // namespace
} // namespace field
