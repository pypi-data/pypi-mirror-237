#include "hip/hip_runtime.h"

#include "constants.hpp"

#include <iostream>
#include <chrono>



__global__ void preprocess_image(uint8_t* data, int* histo, uint8_t* max_color_array, uint8_t* min_color_array,
                                 uint8_t* pixel_found_array, int* start_indices, int* stop_indices) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int pixel_index=start_indices[idx]; pixel_index<stop_indices[idx]; ++pixel_index) {

        bool alpha_valid = data[pixel_index * 4 + 3] >= 125;
        bool not_white = data[pixel_index * 4] <= 250 || data[pixel_index * 4 + 1] <= 250 || data[pixel_index * 4 + 2] <= 250;
        bool mask = alpha_valid && not_white;

        int histo_pixel_index = 0;

        for (int color_index=0; color_index<3; ++color_index) {
            uint8_t color_value = data[pixel_index * 4 + color_index] >> RSHIFT;

            max_color_array[idx * 3 + color_index] = max(max_color_array[idx * 3 + color_index], color_value * mask);
            min_color_array[idx * 3 + color_index] = min(min_color_array[idx * 3 + color_index], color_value * (1 - mask));

            histo_pixel_index += color_value << ((2 - color_index) * SIGBITS);
        }
        atomicAdd(histo + histo_pixel_index, int(mask));
        pixel_found_array[idx] = pixel_found_array[idx] || mask;
    }
}


std::tuple<std::vector<int>, color_t, color_t, bool> get_histo_cuda(uint8_t* data, int pixel_count, int quality) {
    std::vector<int> histo(std::pow(2, 3 * SIGBITS), 0);

    int num_threads = NUM_BLOCKS * THREADS_PER_BLOCK;
    int data_per_thread = std::ceil(double(pixel_count) / double(num_threads));

    std::vector<uint8_t> max_color_array(num_threads * 3, 0);
    std::vector<uint8_t> min_color_array(num_threads * 3, 0);
    std::vector<uint8_t> pixel_found_array(num_threads, 0);
    std::vector<int> start_indices;
    std::vector<int> stop_indices;

    start_indices.reserve(num_threads);
    stop_indices.reserve(num_threads);
    for (int i=0; i<num_threads; ++i) {
      start_indices.push_back(std::min(i * data_per_thread, pixel_count));
      stop_indices.push_back(std::min((i + 1) * data_per_thread, pixel_count));
    }

    if (start_indices.size() != num_threads) {
        std::cout << num_threads << " " << start_indices.size() << std::endl;
        throw std::runtime_error("Bug in preparation of data for cuda");
    }

    uint8_t *cuda_data, *cuda_max_color_array, *cuda_min_color_array;
    int *cuda_histo, *cuda_start_indices, *cuda_stop_indices;
    uint8_t *cuda_pixel_found_array;

    hipMalloc(&cuda_data, pixel_count * 4);
    hipMalloc(&cuda_max_color_array, max_color_array.size());
    hipMalloc(&cuda_min_color_array, min_color_array.size());
    hipMalloc(&cuda_histo, histo.size() * sizeof(int));
    hipMalloc(&cuda_start_indices, start_indices.size() * sizeof(int));
    hipMalloc(&cuda_stop_indices, stop_indices.size() * sizeof(int));
    hipMalloc(&cuda_pixel_found_array, pixel_found_array.size() * sizeof(uint8_t));

    hipMemcpyAsync(cuda_data, data, pixel_count * 4, hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_max_color_array, max_color_array.data(), max_color_array.size(), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_min_color_array, min_color_array.data(), min_color_array.size(), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_histo, histo.data(), histo.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_start_indices, start_indices.data(), start_indices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_stop_indices, stop_indices.data(), stop_indices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_pixel_found_array, pixel_found_array.data(), pixel_found_array.size() * sizeof(uint8_t), hipMemcpyHostToDevice);

    preprocess_image<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(cuda_data, cuda_histo, cuda_max_color_array, cuda_min_color_array,
                                                        cuda_pixel_found_array, cuda_start_indices, cuda_stop_indices);

    hipMemcpyAsync(histo.data(), cuda_histo, histo.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpyAsync(max_color_array.data(), cuda_max_color_array, max_color_array.size(), hipMemcpyDeviceToHost);
    hipMemcpyAsync(min_color_array.data(), cuda_min_color_array, min_color_array.size(), hipMemcpyDeviceToHost);
    hipMemcpyAsync(pixel_found_array.data(), cuda_pixel_found_array, pixel_found_array.size() * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipFree(cuda_data);
    hipFree(cuda_max_color_array);
    hipFree(cuda_min_color_array);
    hipFree(cuda_histo);
    hipFree(cuda_start_indices);
    hipFree(cuda_stop_indices);
    hipFree(cuda_pixel_found_array);

    color_t max_color, min_color;
    bool pixel_found = false;
    for (int thread_index=0; thread_index<num_threads; ++thread_index) {
        for (int color_index=0; color_index<3; ++color_index) {
            max_color[color_index] = std::max(max_color[color_index], max_color_array[thread_index * 3 + color_index]);
            min_color[color_index] = std::min(min_color[color_index], min_color_array[thread_index * 3 + color_index]);
        }
        pixel_found = pixel_found_array[thread_index] || pixel_found;
    }

    return {histo, min_color, max_color, pixel_found};
}
